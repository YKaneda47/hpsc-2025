
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void init_bucket(int* bucket, int range) {
    int i = threadIdx.x;
    if (i < range) {
        bucket[i] = 0;
    }
}

__global__ void count_keys(int* key, int* bucket, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        atomicAdd(&bucket[key[i]], 1);
    }
}

__global__ void sort(int* key, int* bucket, int range) {
    int i = threadIdx.x;
    if (i < range) {
        int count = bucket[i];
        int idx = 0;
        for (int j = 0; j < i; ++j) {
            idx += bucket[j];
        }
        for (int j = 0; j < count; ++j) {
            key[idx + j] = i;
        }
    }
}

int main() {
  int n = 50;
  int range = 5;
  //std::vector<int> key(n);
  int *key;
  int *bucket;
  hipMallocManaged(&key, n*sizeof(int));
  hipMallocManaged(&bucket, range*sizeof(int));

  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");
/*
  std::vector<int> bucket(range); 
  for (int i=0; i<range; i++) {
    bucket[i] = 0;
  }
  for (int i=0; i<n; i++) {
    bucket[key[i]]++;
  }
  for (int i=0, j=0; i<range; i++) {
    for (; bucket[i]>0; bucket[i]--) {
      key[j++] = i;
    }
  }
*/
  init_bucket<<<1, range>>>(bucket, range);
  hipDeviceSynchronize();

  count_keys<<<(n+255) / 256, 256>>>(key, bucket, n);
  hipDeviceSynchronize();

  sort<<<1, range>>>(key, bucket, range);
  hipDeviceSynchronize();

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");

  hipFree(key);
  hipFree(bucket);
}
